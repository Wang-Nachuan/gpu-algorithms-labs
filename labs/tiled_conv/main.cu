#include "hip/hip_runtime.h"
#include "helper.hpp"

#define TILE_SZ_A 32
#define TILE_SZ_B 32
#define TILE_SZ_RATIO (TILE_SZ_A/TILE_SZ_B)

/*
  16, 16: 64.468193ms
  32, 32: 52.173119ms
  64, 32: 54.266911ms
  64, 64：56.390144ms
  128, 32: 76.832321ms
  128, 64: 76.948479ms
  256, 32: 156.062469ms
  256, 64: 165.208069ms
 */

__constant__ float const_k[32 * 1 * 5 * 5];

// Sequential code for the forward path of the convolution layer
// You should not modify this code
static void conv_forward_valid(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y,
                               const shape &ydims) {
  std::fill(Y, Y + ydims.flattened_length(), 0);

  for (auto i : range(0, ydims.num)) {
    for (auto m : range(0, ydims.depth )) {   // for each output feature map
      for (auto h : range(0, ydims.height)) { // for each output element
        for (auto w : range(0, ydims.width )) {
          const auto yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
          for (auto c : range(0, xdims.depth )) {     // sum over all input feature maps
            for (auto p : range(0, wdims.height)) {   // filter height
              for (auto q : range(0, wdims.width )) { // filter width
                const auto xoffset = ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                const auto woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                Y[yoffset] += X[xoffset] * W[woffset];
              }
            }
          }
        }
      }
    }
  }
}

// Baseline GPU kernel code for forward convolution.
// One thread per output index
// You should not modify this kernel as it is used for correctness comparison.
// Instead, define a new one below
__global__ void conv_forward_baseline_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y,
                                    const shape ydims) {


  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t i = gx; i < ydims.num * ydims.depth * ydims.height * ydims.width; i += blockDim.x * gridDim.x) {
    Y[i] = 0.f;
  }

  for (size_t i = gx; i < ydims.num; i += gridDim.x * blockDim.x) {
    for (auto m : range(0, ydims.depth )) { // for each output feature map
      for (auto h : range(0, ydims.height)) { // for each output element
        for (auto w : range(0, ydims.width )) {
          const size_t yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
          for (auto c : range(0, xdims.depth )) {     // sum over all input feature maps
            for (auto p : range(0, wdims.height)) {   // filter height
              for (auto q : range(0, wdims.width )) { // filter width
                const size_t xoffset = ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                const size_t woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                Y[yoffset] += X[xoffset] * W[woffset];
              }
            }
          }
        }
      }
    }
  }
}

// Host code to configure baseline GPU kernel
static void convlayer_gpu_baseline(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y,
  const shape &ydims) {

  dim3 dimGrid(1);
  dim3 dimBlock(32);

  conv_forward_baseline_kernel<<<dimGrid, dimBlock>>>(X, xdims, W, wdims, Y, ydims);
  THROW_IF_ERROR(hipGetLastError());

}

// Implement your optimized kernel here.
// Make any modifications you wish.
// Don't forget to modify the host code below, if needed!
__global__ void conv_forward_opt_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y,
  const shape ydims) {

  //@@ YOUR CODE HERE!
  
  // Virtualize input/weight as matrix
  int b = blockIdx.z;
  int M = ydims.depth;
  int C1 = xdims.depth;
  int H = xdims.height;
  int W1 = xdims.width;
  int K = wdims.height;
  int H_out = H - K + 1;
  int W_out = W1 - K + 1;
  int K_size = K * K;

  #define C(ri, ci) Y[(b) * (M * H_out * W_out) + (ri) * (H_out * W_out) + ((ci) / W_out) * (W_out) + ((ci) % W_out)]
  #define B(ri, ci) X[(b) * (C1 * H * W1) + ((ri) / K_size) * (H * W1) + (((ri) % K_size) / K + ((ci) / W_out)) * (W1) + (((ri) % K_size) % K + ((ci) % W_out))]
  #define A(ri, ci) const_k[(ri) * (C1 * K * K) + (ci)]

  int m = M;
  int k = K_size * C1;
  int n = H_out * W_out;

  // Do the normal tiled matrix multiplication
  __shared__ float B_ds[TILE_SZ_RATIO][TILE_SZ_B];
  int ti = threadIdx.x;     // Thread index
  int row = blockIdx.x * TILE_SZ_A + ti;
  int col = blockIdx.y * TILE_SZ_B;
  float out[TILE_SZ_B] = {0};

  for (int tn = 0; tn < ceil((float) k / TILE_SZ_RATIO); tn++) {
    // Load shared memory
    int rowS = ti / TILE_SZ_B;
    int colS = ti % TILE_SZ_B;
    int rowB = tn * TILE_SZ_RATIO + rowS;
    int colB = col + colS;
    if (rowB < k && colB < n) {
      B_ds[rowS][colS] = B(rowB, colB);
    } else {
      B_ds[rowS][colS] = 0;
    }
    __syncthreads();

    // Load register & Accumulate result
    for (int i = 0; i < TILE_SZ_RATIO; i++) {
      if (row < m && tn * TILE_SZ_RATIO < k) {
        float Ai = A(row, tn * TILE_SZ_RATIO + i);
        #pragma unroll
        for (int j = 0; j < TILE_SZ_B; j++) {
          out[j] += Ai * B_ds[i][j];
        }
      }
    }
    __syncthreads();
  }

  // Write result
  if (row < m) {
    #pragma unroll
    for (int j = 0; j < TILE_SZ_B; j++) {
      if (col + j < n) {
        C(row, col + j) = out[j];
      }
    }
  }

  #undef A
  #undef B
  #undef C
}

// Host code to configure baseline GPU kernel
static void convlayer_gpu_opt(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y,
  const shape &ydims) {

  // Modify this code to configure your optimized kernel.
  //@@ YOUR CODE HERE!!!
  int H_out = xdims.height - wdims.height + 1;
  int W_out = xdims.width - wdims.width + 1;
  int mtx_numRow = ydims.depth;
  int mtx_numCol = H_out * W_out;
  dim3 dimGrid(ceil((float) mtx_numRow/TILE_SZ_A), ceil((float) mtx_numCol/TILE_SZ_B), ydims.num);
  // dim3 dimGrid(ceil((float) mtx_numCol/TILE_SZ_A), ceil((float) mtx_numRow/TILE_SZ_B), ydims.num);
  dim3 dimBlock(TILE_SZ_A, 1, 1);
  conv_forward_opt_kernel<<<dimGrid, dimBlock>>>(X, xdims, W, wdims, Y, ydims);
  THROW_IF_ERROR(hipGetLastError());

}


static int eval(const shape wDims, const shape xDims, bool doVerify) {

  // Generate model
  const auto conf_info = std::string("conv[wDims:") + std::to_string(wDims.num) + "," +
                                                      std::to_string(wDims.depth) + "," +
                                                      std::to_string(wDims.height) + "," +
                                                      std::to_string(wDims.width) +
                                                      " xDims:" + std::to_string(xDims.num) + "," +
                                                      std::to_string(xDims.depth) + "," +
                                                      std::to_string(xDims.height) + "," +
                                                      std::to_string(xDims.width) + "]";
  INFO("Running "  << conf_info);

  // Generate convolution weights
  float *hostW = allocate<float>(wDims);
  generate_convfilters(hostW, wDims);

  // generate input feature map
  float *hostX = allocate<float>(xDims);
  generate_data(hostX, xDims);

  // generate output feature map for verification
  const shape ydims = {xDims.num, wDims.num, (xDims.height - wDims.height + 1),
      (xDims.width - wDims.width + 1)};
  INFO("Allocating output tensor [" << ydims.num << "," << ydims.depth << "," << ydims.height << "," << ydims.width << "]");
  float *hostY = allocate<float>(ydims);
  float *expected = allocate<float>(ydims);
  generate_data(hostY, ydims);


  const size_t wByteCount = wDims.flattened_length() * sizeof(float);
  const size_t xByteCount = xDims.flattened_length() * sizeof(float);
  const size_t yByteCount = ydims.flattened_length() * sizeof(float);

  float *deviceW = nullptr, *deviceX = nullptr, *deviceY = nullptr;
  timer_start("Allocating GPU memory.");
  THROW_IF_ERROR(hipMalloc((void **)&deviceW, wByteCount));
  THROW_IF_ERROR(hipMalloc((void **)&deviceX, xByteCount));
  THROW_IF_ERROR(hipMalloc((void **)&deviceY, yByteCount));
  timer_stop();


  timer_start("Copying inputs to the GPU.");
  THROW_IF_ERROR(hipMemcpy(deviceW, hostW, wByteCount, hipMemcpyDefault));
  THROW_IF_ERROR(hipMemcpy(deviceX, hostX, xByteCount, hipMemcpyDefault));
  hipMemcpyToSymbol(HIP_SYMBOL(const_k), hostW, wByteCount);
  timer_stop();

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  timer_start("Performing GPU convlayer");
  convlayer_gpu_opt(deviceX, xDims, deviceW, wDims, deviceY, ydims);
  THROW_IF_ERROR(hipDeviceSynchronize());
  timer_stop();

  // verify with provided implementation
  if (doVerify) {
    timer_start("Copying output to the CPU");
    THROW_IF_ERROR(hipMemcpy(hostY, deviceY, yByteCount, hipMemcpyDefault));
    timer_stop();

    convlayer_gpu_baseline(deviceX, xDims, deviceW, wDims, deviceY, ydims);
    THROW_IF_ERROR(hipDeviceSynchronize());
    THROW_IF_ERROR(hipMemcpy(expected, deviceY, yByteCount, hipMemcpyDefault));
    // conv_forward_valid(hostX, xDims, hostW, wDims, expected, ydims);
    verify(expected, hostY, ydims);
  }

  THROW_IF_ERROR(hipFree(deviceW));
  THROW_IF_ERROR(hipFree(deviceX));
  THROW_IF_ERROR(hipFree(deviceY));
  free(hostW);
  free(hostX);
  free(hostY);
  free(expected);

  return 0;
}



TEST_CASE("Convlayer", "[convlayer]") {
#if 0
  // test five times in case code errors depend on data
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32,1,5,5}, {20,1,28,28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32,1,5,5}, {20,1,28,28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32,1,5,5}, {20,1,28,28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32,1,5,5}, {20,1,28,28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32,1,5,5}, {20,1,28,28}, true);
  }
#else
  SECTION("[wDims:32,1,5,5 xDims:50000,1,28,28]") {
    eval({32,1,5,5}, {50000,1,28,28}, false);
  }
#endif
}
